#include "hip/hip_runtime.h"
/*
  convolve_iir_gpu function for convolving an image horizontally using pure cuda

*/
#include "fastfilters.hxx"

#define N_REGISTERS_CONVOLVE_ROW 40 // determine with -Xptxas -v compiler flags


__global__ void convolve_row( float* input, float* output, int M, int N, int n_border, 
			      float* d, float* causal, float* anticausal )
{
  int dim = blockIdx.x*blockDim.x + threadIdx.x;
  if (dim < M) { 
     // original row code 
  }
}


void convolve_iir_gpu( const float* input, float* output, const unsigned int n_cols, const unsigned int n_rows,
                       const fastfilters::iir::Coefficients &coefs)
{
  hipSetDevice(0);
  float *in, *out, *d, *causal, *anticausal; 
  unsigned int n_blocks, n_threads_per_block, coefs_size, data_size;  
  hipDeviceProp_t prop;

  data_size = n_cols * n_rows * sizeof(float);
  coefs_size = 4 * sizeof(float);
  
  hipMalloc( &in, data_size );
  hipMalloc( &out, data_size );
  hipMalloc( &d, coefs_size );
  hipMalloc( &causal, coefs_size );
  hipMalloc( &anticausal, coefs_size );

  hipMemcpy( in, input, data_size, hipMemcpyHostToDevice );

  hipMemcpy( d, coefs.d.data(), coefs_size, hipMemcpyHostToDevice );
  hipMemcpy( causal, coefs.n_causal.data(), coefs_size, hipMemcpyHostToDevice );
  hipMemcpy( anticausal, coefs.n_anticausal.data(), coefs_size, hipMemcpyHostToDevice );

  hipGetDeviceProperties( &prop, 0 );
  n_blocks = ceil( N_REGISTERS_CONVOLVE_ROW * n_rows / (float) prop.regsPerBlock ); 
  n_threads_per_block = ceil( n_rows/ (float) n_blocks );
  
  convolve_row<<< n_blocks, n_threads_per_block >>>
    ( in, out, n_rows, n_cols, coefs.n_border, d, causal, anticausal );

  hipMemcpy( output, out, data_size, hipMemcpyDeviceToHost );
  
  hipFree( in );
  hipFree( out );
  hipFree( d );
  hipFree( causal );
  hipFree( anticausal );
}

