/*
  Program to time functions convolving images using the thrust library 
  Prints time in nanoseconds

  Usage: ./iirfilter-thrust pngfile
*/
#include "fastfilters.hxx"
#include "image.hxx"

#include <string>
#include <chrono>
#include <stdlib.h>
#include <stdio.h>

#include <thrust/functional.h>
#include <thrust/tabulate.h>
#include <thrust/device_vector.h>
#include <thrust/tuple.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/random.h>

typedef thrust::tuple<float, float, float, float> FloatTuple;


struct RandGen
{
    RandGen() {}

    __host__ __device__
    float operator () (int idx)
    { 
       thrust::default_random_engine randEng;
       thrust::uniform_real_distribution<float> uniDist(-3.2, 3.2);
       randEng.discard(idx);
       return uniDist(randEng);
    }
};

struct convolve_4
{
  const FloatTuple d;
  const FloatTuple causal;

  convolve_4( const FloatTuple _d, const FloatTuple _causal ) : d(_d), causal(_causal) {}

   template <typename Tuple, typename Tuple2>
  __host__ __device__
  Tuple operator()(const Tuple &y, const Tuple2 &x)
  {
    using thrust::get;
    float sum =  get<0>(causal) * get<0>(x)
               + get<1>(causal) * get<1>(x)
               + get<2>(causal) * get<2>(x)
               + get<3>(causal) * get<3>(x)
               - get<0>(d) * get<0>(y) 
               - get<1>(d) * get<1>(y)
               - get<2>(d) * get<2>(y) 
               - get<3>(d) * get<3>(y);
    return thrust::make_tuple(sum, get<0>(y), get<1>(y), get<2>(y));
  }
};

struct addfirsts : public thrust::binary_function<FloatTuple, FloatTuple, float>
{
  __host__ __device__
  float operator()( const FloatTuple& x, const FloatTuple& y ) const
  {
    return thrust::get<0>(x) + thrust::get<0>(y);
  }
};

struct row_idx : public thrust::unary_function<int, int>
{
  const int width;
  
  row_idx(int _width) : width(_width)  {}
  
  __host__ __device__
  int operator()( const int idx ) const
  {
    return (idx) / width;
  }
};

void convolve_iir_gpu(const float* input, float* output, int n_cols, int n_rows,
                       fastfilters::iir::Coefficients &coefs )
{
  hipSetDevice(0);
  unsigned int n_pixels, n_border;

  n_pixels = n_cols * n_rows;
  n_border = 3; // since window size = 4

  FloatTuple d(coefs.d[0],coefs.d[1], coefs.d[2], coefs.d[3]);
  FloatTuple causal(coefs.n_causal[0],coefs.n_causal[1], coefs.n_causal[2], coefs.n_causal[3]);
  FloatTuple anticausal(coefs.n_anticausal[0],coefs.n_anticausal[1], coefs.n_anticausal[2], coefs.n_anticausal[3]);

  thrust::device_vector<float> in(n_pixels+2*n_border); // left and right border 
  thrust::device_vector<float> out(n_pixels);
  
  // pad border with zeros
  thrust::fill(in.begin(), in.begin()+n_border, 0.0);
  thrust::copy_n(input, n_pixels, in.begin()+n_border);
  thrust::fill(in.begin()+n_border+n_pixels, in.end(), 0.0);

  FloatTuple zero(0,0,0,0);
  thrust::device_vector<FloatTuple> forth(n_pixels,zero);
  thrust::device_vector<FloatTuple> back(n_pixels,zero);

  thrust::device_vector<int> key(n_pixels);
  thrust::tabulate(key.begin(), key.end(), row_idx(n_cols));
  thrust::equal_to<int> eq;

  exclusive_scan_by_key(key.begin(), 
                        key.end(),
                        thrust::make_zip_iterator(thrust::make_tuple(in.begin()+3, in.begin()+2, in.begin()+1, in.begin()))+1,
                        forth.begin(), 
                        thrust::make_tuple(coefs.d[0]*in[3],0,0,0),
                        eq,
                        convolve_4(d,causal));
  
  auto in_rev = in.rbegin();
  thrust::exclusive_scan_by_key
    (key.begin(),
     key.end(),
     thrust::make_zip_iterator (thrust::make_tuple(in_rev+3, in_rev+2, in_rev+1, in_rev)),
     back.begin(),
     thrust::make_tuple(coefs.d[0]*in_rev[3], 0.0, 0.0, 0.0),
     eq,
     convolve_4(d, anticausal));
    
  auto back_rev = back.rbegin();
  thrust::transform(forth.begin(), forth.end(), back_rev, out.begin(), addfirsts());
  thrust::copy(out.begin(), out.end(), output);

}


int main( int argc, char* argv[] )
{
  // check commandline parameters
  if( argc != 2 )
    {
      std::cout << "Usage: ./iirfilter-cuda pngfile" << std::endl;
      exit(0);
    }

  // for testing - TODO!
  int n = 10;
  thrust::device_vector<float> d_t1(n);
  thrust::tabulate(d_t1.begin(), d_t1.end(), RandGen());
  
  // determine file names
  std::string infile = argv[1];
  std::string outfile = infile.substr( 0, infile.length() - 4 ) + "_blurred_thrust.png";

  // initialize input parameters for function
  Image input = Image( infile );
  fastfilters::iir::Coefficients coefs( 5.0, 0 );
  float output_data[input.width()*input.height()];
  
  // time CUDA function
  auto begin = std::chrono::high_resolution_clock::now();
  hipDeviceSynchronize();
  convolve_iir_gpu( input.data(), output_data, input.width(), input.height(), coefs);
  hipDeviceSynchronize();
  auto end = std::chrono::high_resolution_clock::now();
  printf("%d", (end-begin).count()); // time in nanoseconds
  Image output = Image( output_data, input.width(), input.height() );
  output.write( outfile );

  return 0;
}
