#include "hip/hip_runtime.h"
/*
  Program to time functions convolving images using cuda
  Prints time in nanoseconds

  Usage: ./iirfilter-cuda pngfile
*/

#include "fastfilters.hxx"
#include "image.hxx"

#include <iostream>
#include <string>
#include <chrono>
#include <stdlib.h>

__global__ void convolve_row( float* input, float* output, int N, const Coefficients &coefs)
{
  int dim = threadIdx.x;
    
  const float *cur_line = input + dim * N;
  float *cur_output = output + dim * N;
  float xtmp[4];
  float ytmp[4];
  float tmpbfr[N];

  for (unsigned int i = 0; i < 4; ++i)
    xtmp[i] = ytmp[i] = 0.0;

  // left border
  for (unsigned int i = 0; i < coefs.n_border; ++i) {
    float sum = 0.0;

    xtmp[0] = cur_line[coefs.n_border - i];
    for (unsigned int j = 0; j < 4; ++j)
      sum += coefs.n_causal[j] * xtmp[j];
    for (unsigned int j = 0; j < 4; ++j)
      sum -= coefs.d[j] * ytmp[j];
    for (unsigned int j = 3; j > 0; --j) {
      xtmp[j] = xtmp[j - 1];
      ytmp[j] = ytmp[j - 1];
    }

    ytmp[0] = sum;
  }

  // causal pass
  for (unsigned int i = 0; i < N; ++i) {
    float sum = 0.0;

    xtmp[0] = cur_line[i];
    for (unsigned int j = 0; j < 4; ++j)
      sum += coefs.n_causal[j] * xtmp[j];
    for (unsigned int j = 0; j < 4; ++j)
      sum -= coefs.d[j] * ytmp[j];
    for (unsigned int j = 3; j > 0; --j) {
      xtmp[j] = xtmp[j - 1];
      ytmp[j] = ytmp[j - 1];
    }

    tmpbfr[i] = sum;
    ytmp[0] = sum;
  }

  // reset variables for anti-causal pass
  for (unsigned int i = 0; i < 4; ++i)
    xtmp[i] = ytmp[i] = 0.0;

  // right border
  for (int i = coefs.n_border; i > 0; --i) {
    float sum = 0.0;

    for (unsigned int j = 0; j < 4; ++j)
      sum += coefs.n_anticausal[j] * xtmp[j];
    for (unsigned int j = 0; j < 4; ++j)
      sum -= coefs.d[j] * ytmp[j];
    for (unsigned int j = 3; j > 0; --j) {
      xtmp[j] = xtmp[j - 1];
      ytmp[j] = ytmp[j - 1];
    }

    xtmp[0] = cur_line[N - i];
    ytmp[0] = sum;
  }

  // anti-causal pass
  for (int i = N - 1; i >= 0; --i) {
    float sum = 0.0;

    for (unsigned int j = 0; j < 4; ++j)
      sum += coefs.n_anticausal[j] * xtmp[j];
    for (unsigned int j = 0; j < 4; ++j)
      sum -= coefs.d[j] * ytmp[j];
    for (unsigned int j = 3; j > 0; --j) {
      xtmp[j] = xtmp[j - 1];
      ytmp[j] = ytmp[j - 1];
    }

    xtmp[0] = cur_line[i];
    ytmp[0] = sum;
    cur_output[i] = tmpbfr[i] + sum;
  }
}


void convolve_iir_gpu( float* input, float* output, int n_cols, int n_rows,
                       const Coefficients &coefs)
{
  convolve_row<< 1, n_rows >>(input, output, n_cols);
}


int main( int argc, char* argv[] )
{
  // check commandline parameters
  if( argc != 2 )
    {
      std::cout << "Usage: ./iirfilter-cuda pngfile" << std::endl;
      exit(0);
    }

  // determine file names
  std::string infile = argv[1];
  std::string outfile = infile.substr( 0, infile.length() - 4 ) + "_blurred_cuda.png";

  // initialize input parameters for function
  Image input = Image(infile);
  fastfilters::iir::Coefficients coefs( 5.0, 0 );
  float output_data[input.width()*input.height()];

  // time CUDA function
  auto begin = std::chrono::high_resolution_clock::now();
  int N = input.width() * input.height();
  size_t size = N * sizeof(float);
  hipDeviceSynchronize();
  float* d_in;
  hipMalloc( &d_in, size );
  float* d_out;
  hipMalloc( &d_out, size );
  hipMemcpy( d_in, input.data(), size );
  convolve_iir_gpu( d_in, d_out, input.width(), input.height(), coefs);
  hipMemcpy( output_data, d_out, size );
  hipFree(d_in);
  hipFree(d_out);
  hipDeviceSynchronize();

  auto end = std::chrono::high_resolution_clock::now();
  std::cout << std::chrono::duration_cast<std::chrono::nanoseconds>(end-begin).count() << std::endl;

  // write result back to output file
  Image output = Image( output_data, input.width(), input.height() );
  output.write( outfile );

  return 0;
}
